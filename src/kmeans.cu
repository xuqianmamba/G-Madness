#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <iostream>
#include <cassert>
 
#include "kmeans.h"
 
using namespace std;
 
const int MAX_CHAR_PER_LINE = 1024;
 
class KMEANS
{
private:
	int numClusters;
	int numCoords;
	int numObjs;
	int *membership;//[numObjs]
	char *filename; 
	float **objects;//[numObjs][numCoords] data objects
	float **clusters;//[numClusters][unmCoords] cluster center
	float threshold;
	int loop_iterations;
 
public:
	KMEANS(int k);
	void file_read(char *fn);
	void file_write();
	void cuda_kmeans();
	inline int nextPowerOfTwo(int n);
	void free_memory();
	virtual ~KMEANS();
};
 
KMEANS::~KMEANS()
{
	free(membership);
	free(clusters[0]);
	free(clusters);
	free(objects[0]);
	free(objects);
}
 
KMEANS::KMEANS(int k)
{
	threshold = 0.001;
	numObjs = 0;
	numCoords = 0;
	numClusters = k;
	filename = NULL;
	loop_iterations = 0;
}
 
void KMEANS::file_write()
{
	FILE *fptr;
	char outFileName[1024];
 
	//output:the coordinates of the cluster centres
	sprintf(outFileName,"%s.cluster_centres",filename);
	printf("Writingcoordinates of K=%d cluster centers to file \"%s\"\n",numClusters,outFileName);
	fptr = fopen(outFileName,"w");
	for(int i=0;i<numClusters;i++)
	{
		fprintf(fptr,"%d ",i)	;
		for(int j=0;j<numCoords;j++)
			fprintf(fptr,"%f ",clusters[i][j]);
		fprintf(fptr,"\n");
	}
	fclose(fptr);
 
	//output:the closest cluster centre to each of the data points
	sprintf(outFileName,"%s.membership",filename);
	printf("writing membership of N=%d data objects to file \"%s\" \n",numObjs,outFileName);
	fptr = fopen(outFileName,"w");
	for(int i=0;i<numObjs;i++)
	{
		fprintf(fptr,"%d %d\n",i,membership[i])	;
	}
	fclose(fptr);
}
 
inline int KMEANS::nextPowerOfTwo(int n)
{
	n--;
	n = n >> 1 | n;
	n = n >> 2 | n;
	n = n >> 4 | n;
	n = n >> 8 | n;
	n = n >> 16 | n;
	//n = n >> 32 | n; // for 64-bit ints
	return ++n;
}
 
__host__ __device__ inline static 
float euclid_dist_2(int numCoords,int numObjs,int numClusters,float *objects,float *clusters,int objectId,int clusterId)
{
	int i;
	float ans = 0;
	for( i=0;i<numCoords;i++ )
	{
		ans += ( objects[numObjs * i + objectId] - clusters[numClusters*i + clusterId] ) *
			   ( objects[numObjs * i + objectId] - clusters[numClusters*i + clusterId] ) ;
	}
	return ans;
}
 
/*
* numIntermediates:The actual number of intermediates
* numIntermediates2:The next power of two
*/
__global__ static void compute_delta(int *deviceIntermediates,int numIntermediates,	int numIntermediates2)
{
	extern __shared__ unsigned int intermediates[];
 
	intermediates[threadIdx.x] = (threadIdx.x < numIntermediates) ? deviceIntermediates[threadIdx.x] : 0 ;
	__syncthreads();
 
	//numIntermediates2 *must* be a power of two!
	for(unsigned int s = numIntermediates2 /2 ; s > 0 ; s>>=1)
	{
		if(threadIdx.x < s)	
		{
			intermediates[threadIdx.x] += intermediates[threadIdx.x + s];	
		}
		__syncthreads();
	}
	if(threadIdx.x == 0)
	{
		deviceIntermediates[0] = intermediates[0];
	}
}
 
/*
* objects:[numCoords][numObjs]
* deviceClusters:[numCoords][numClusters]
* membership:[numObjs]
*/
__global__ static void find_nearest_cluster(int numCoords,int numObjs,int numClusters,float *objects, float *deviceClusters,int *membership ,int *intermediates)
{
	extern __shared__ char sharedMemory[];
	unsigned char *membershipChanged = (unsigned char *)sharedMemory;
	float *clusters = deviceClusters;
 
	membershipChanged[threadIdx.x] = 0;
 
	int objectId = blockDim.x * blockIdx.x + threadIdx.x;
	if( objectId < numObjs )
	{
		int index;
		float dist,min_dist;
		/*find the cluster id that has min distance to object*/
		index = 0;
		min_dist = euclid_dist_2(numCoords,numObjs,numClusters,objects,clusters,objectId,0);
		
		for(int i=0;i<numClusters;i++)
		{
			dist = euclid_dist_2(numCoords,numObjs,numClusters,objects,clusters,objectId,i)	;
			/* no need square root */
			if( dist < min_dist )
			{
				min_dist = dist;
				index = i;
			}
		}
 
		if( membership[objectId]!=index )
		{
			membershipChanged[threadIdx.x] = 1;	
		}
		//assign the membership to object objectId
		membership[objectId] = index;
 
		__syncthreads(); //for membershipChanged[]
 
#if 1
		//blockDim.x *must* be a power of two!
		for(unsigned int s = blockDim.x / 2; s > 0 ;s>>=1)
		{
			if(threadIdx.x < s)	
			{
				membershipChanged[threadIdx.x] += membershipChanged[threadIdx.x + s];//calculate all changed values and save result to membershipChanged[0]
			}
			__syncthreads();
		}
		if(threadIdx.x == 0)
		{
			intermediates[blockIdx.x] = membershipChanged[0];
		}
#endif
	}
}//find_nearest_cluster
 
void KMEANS::cuda_kmeans()
{
	int index,loop = 0;
	int *newClusterSize;//[numClusters]:no.objects assigned in each new cluster
	float delta; //% of objects changes their clusters
	float **dimObjects;//[numCoords][numObjs]
	float **dimClusters;
	float **newClusters;//[numCoords][numClusters]
 
	float *deviceObjects; //[numCoords][numObjs]
	float *deviceClusters; //[numCoords][numclusters]
	int *deviceMembership;
	int *deviceIntermediates;
 
	//Copy objects given in [numObjs][numCoords] layout to new [numCoords][numObjs] layout
	malloc2D(dimObjects,numCoords,numObjs,float);
	for(int i=0;i<numCoords;i++)
	{
		for(int j=0;j<numObjs;j++)
		{
			dimObjects[i][j] = objects[j][i];	
		}
	}
	//pick first numClusters elements of objects[] as initial cluster centers
	malloc2D(dimClusters, numCoords, numClusters,float);
	for(int i=0;i<numCoords;i++)
	{
		for(int j=0;j<numClusters;j++)
		{
			dimClusters[i][j] = dimObjects[i][j];
		}
	}
	newClusterSize = new int[numClusters];
	assert(newClusterSize!=NULL);
	malloc2D(newClusters,numCoords,numClusters,float);
	memset(newClusters[0],0,numCoords * numClusters * sizeof(float) );
	
	//To support reduction,numThreadsPerClusterBlock *must* be a power of two, and it *must* be no larger than the number of bits that will fit into an unsigned char ,the type used to keep track of membership changes in the kernel.
	const unsigned int numThreadsPerClusterBlock = 32;
	const unsigned int numClusterBlocks = (numObjs + numThreadsPerClusterBlock -1)/numThreadsPerClusterBlock;
	const unsigned int numReductionThreads = nextPowerOfTwo(numClusterBlocks);
 
	const unsigned int clusterBlockSharedDataSize = numThreadsPerClusterBlock * sizeof(unsigned char);
 
	const unsigned int reductionBlockSharedDataSize = numReductionThreads * sizeof(unsigned int);
 
	hipMalloc(&deviceObjects,numObjs*numCoords*sizeof(float));
	hipMalloc(&deviceClusters,numClusters*numCoords*sizeof(float));
	hipMalloc(&deviceMembership,numObjs*sizeof(int));
	hipMalloc(&deviceIntermediates,numReductionThreads*sizeof(unsigned int));
 
	hipMemcpy(deviceObjects,dimObjects[0],numObjs*numCoords*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(deviceMembership,membership,numObjs*sizeof(int),hipMemcpyHostToDevice);
 
	do
	{
		hipMemcpy(deviceClusters,dimClusters[0],numClusters*numCoords*sizeof(float),hipMemcpyHostToDevice);
 
		find_nearest_cluster<<<numClusterBlocks,numThreadsPerClusterBlock,clusterBlockSharedDataSize>>>(numCoords,numObjs,numClusters,deviceObjects,deviceClusters,deviceMembership,deviceIntermediates);
 
		hipDeviceSynchronize();
 
		compute_delta<<<1,numReductionThreads,reductionBlockSharedDataSize>>>(deviceIntermediates,numClusterBlocks,numReductionThreads);
 
		hipDeviceSynchronize();
		
		int d;
		hipMemcpy(&d,deviceIntermediates,sizeof(int),hipMemcpyDeviceToHost);
		delta = (float)d;
 
		hipMemcpy(membership,deviceMembership,numObjs*sizeof(int),hipMemcpyDeviceToHost);
		
		for(int i=0;i<numObjs;i++)
		{
			//find the array index of nestest 
			index = membership[i];
			//update new cluster centers:sum of objects located within
			newClusterSize[index]++;
			for(int j=0;j<numCoords;j++)
			{
				newClusters[j][index] += objects[i][j];
			}
		}
		//average the sum and replace old cluster centers with newClusters 
		for(int i=0;i<numClusters;i++)
		{
			for(int j=0;j<numCoords;j++)
			{
				if(newClusterSize[i] > 0)	
					dimClusters[j][i] = newClusters[j][i]/newClusterSize[i];
				newClusters[j][i] = 0.0;//set back to 0
			}
			newClusterSize[i] = 0 ; //set back to 0
		}
		delta /= numObjs;
	}while( delta > threshold && loop++ < 500 );
 
	loop_iterations = loop + 1;
	
	malloc2D(clusters,numClusters,numCoords,float);
	for(int i=0;i<numClusters;i++)
	{
		for(int j=0;j<numCoords;j++)
		{
			clusters[i][j] = dimClusters[j][i];
		}
	}
 
	hipFree(deviceObjects)	;
	hipFree(deviceClusters);
	hipFree(deviceMembership);
	hipFree(deviceMembership);
 
	free(dimObjects[0]);
	free(dimObjects);
	free(dimClusters[0]);
	free(dimClusters);
	free(newClusters[0]);
	free(newClusters);
	free(newClusterSize);
}
 
void KMEANS::file_read(char *fn)
{
 
	FILE *infile;
	char *line = new char[MAX_CHAR_PER_LINE];
	int lineLen = MAX_CHAR_PER_LINE;
 
	filename = fn;
	infile = fopen(filename,"r");
	assert(infile!=NULL);
	/*find the number of objects*/	
	while( fgets(line,lineLen,infile) )
	{
		numObjs++;	
	}
 
	/*find the dimension of each object*/
	rewind(infile);
	while( fgets(line,lineLen,infile)!=NULL )
	{
		if( strtok(line," \t\n")!=0 )	
		{
			while( strtok(NULL," \t\n") )	
				numCoords++;
			break;
		}
	}
 
	/*allocate space for object[][] and read all objcet*/
	rewind(infile);
	objects = new float*[numObjs];
	for(int i=0;i<numObjs;i++)
	{
		objects[i] = new float[numCoords];
	}
	int i=0;
	/*read all object*/
	while( fgets(line,lineLen,infile)!=NULL )
	{
		if( strtok(line," \t\n") ==NULL ) continue;
		for(int j=0;j<numCoords;j++)
		{
			objects[i][j] = atof( strtok(NULL," ,\t\n") )	;
		}
		i++;
	}
	
	/* membership: the cluster id for each data object */
	membership = new int[numObjs];
	assert(membership!=NULL);
	for(int i=0;i<numObjs;i++)
		membership[i] = -1;
	
}
 
int main(int argc,char *argv[])
{
	KMEANS kmeans(atoi(argv[1]));
	kmeans.file_read(argv[2]);
	kmeans.cuda_kmeans();
	kmeans.file_write();
	return 0;
}